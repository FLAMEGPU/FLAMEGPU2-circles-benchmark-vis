#include "hip/hip_runtime.h"
#include <algorithm>

#include "flamegpu/flamegpu.h"
#include "common.cuh"

namespace {

FLAMEGPU_AGENT_FUNCTION(output_message, flamegpu::MessageNone, flamegpu::MessageSpatial3D) {
    FLAMEGPU->message_out.setVariable<int>("id", FLAMEGPU->getVariable<int>("id"));
    FLAMEGPU->message_out.setLocation(
        FLAMEGPU->getVariable<float>("x"),
        FLAMEGPU->getVariable<float>("y"),
        FLAMEGPU->getVariable<float>("z"));
    return flamegpu::ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(move, flamegpu::MessageSpatial3D, flamegpu::MessageNone) {
    const int ID = FLAMEGPU->getVariable<int>("id");
    const float REPULSE_FACTOR = FLAMEGPU->environment.getProperty<float>("repulse");
    const float RADIUS = FLAMEGPU->message_in.radius();
    float fx = 0.0;
    float fy = 0.0;
    float fz = 0.0;
    const float x1 = FLAMEGPU->getVariable<float>("x");
    const float y1 = FLAMEGPU->getVariable<float>("y");
    const float z1 = FLAMEGPU->getVariable<float>("z");
    int count = 0;
    int messageCount = 0;
    for (const auto &message : FLAMEGPU->message_in(x1, y1, z1)) {
        if (message.getVariable<int>("id") != ID) {
            const float x2 = message.getVariable<float>("x");
            const float y2 = message.getVariable<float>("y");
            const float z2 = message.getVariable<float>("z");
            float x21 = x2 - x1;
            float y21 = y2 - y1;
            float z21 = z2 - z1;
            const float separation = cbrt(x21*x21 + y21*y21 + z21*z21);
            if (separation < RADIUS && separation > 0.0f) {
                float k = sinf((separation / RADIUS)*3.141*-2)*REPULSE_FACTOR;
                // Normalise without recalculating separation
                x21 /= separation;
                y21 /= separation;
                z21 /= separation;
                fx += k * x21;
                fy += k * y21;
                fz += k * z21;
                count++;
            }
        }
        messageCount++;
    }
    fx /= count > 0 ? count : 1;
    fy /= count > 0 ? count : 1;
    fz /= count > 0 ? count : 1;
    FLAMEGPU->setVariable<float>("x", x1 + fx);
    FLAMEGPU->setVariable<float>("y", y1 + fy);
    FLAMEGPU->setVariable<float>("z", z1 + fz);
    FLAMEGPU->setVariable<float>("drift", cbrt(fx*fx + fy*fy + fz*fz));
    float totalMessageCount = FLAMEGPU->getVariable<float>("totalMessageCount");
    FLAMEGPU->setVariable<float>("totalMessageCount", totalMessageCount + messageCount);
    FLAMEGPU->setVariable<float>("stepMessageCount", messageCount);
    return flamegpu::ALIVE;
}

}  // namespace

// Run an individual simulation, using 
void run_circles_spatial3D(const RunSimulationInputs runInputs, RunSimulationOutputs &runOutputs){
    flamegpu::ModelDescription model("circles_spatial3D");
    // Calculate environment bounds.
    const float ENV_WIDTH = runInputs.ENV_WIDTH;
    const float ENV_MIN = -0.5f * ENV_WIDTH;
    const float ENV_MAX = ENV_MIN + ENV_WIDTH;
    // Compute the actual density and return it.
    runOutputs.agentDensity = runInputs.AGENT_COUNT / (ENV_WIDTH * ENV_WIDTH * ENV_WIDTH);
    {   // Location message
        flamegpu::MessageSpatial3D::Description &message = model.newMessage<flamegpu::MessageSpatial3D>("location");
        message.newVariable<int>("id");
        message.setRadius(runInputs.COMM_RADIUS);
        message.setMin(ENV_MIN, ENV_MIN, ENV_MIN);
        message.setMax(ENV_MAX, ENV_MAX, ENV_MAX);
    }
    {   // Circle agent
        flamegpu::AgentDescription &agent = model.newAgent("Circle");
        agent.newVariable<int>("id");
        agent.newVariable<float>("x");
        agent.newVariable<float>("y");
        agent.newVariable<float>("z");
        agent.newVariable<float>("totalMessageCount", 0.f);
        agent.newVariable<float>("stepMessageCount", 0.f);
        agent.newVariable<float>("drift");  // Store the distance moved here, for validation
        agent.newFunction("output_message", output_message).setMessageOutput("location");
        agent.newFunction("move", move).setMessageInput("location");
    }

    // Global environment variables.
    {
        flamegpu::EnvironmentDescription &env = model.Environment();
        env.newProperty("repulse", ENV_REPULSE);
    }

    // Organise the model. 

    {   // Layer #1
        flamegpu::LayerDescription &layer = model.newLayer();
        layer.addAgentFunction(output_message);
    }
    {   // Layer #2
        flamegpu::LayerDescription &layer = model.newLayer();
        layer.addAgentFunction(move);
    }

    // create step logging for drift and message data
    flamegpu::StepLoggingConfig step_log_cfg(model);
    {
        step_log_cfg.setFrequency(1);
        step_log_cfg.agent("Circle").logMean<float>("drift");
        step_log_cfg.agent("Circle").logMean<float>("stepMessageCount");
    }

    // create exit log
    flamegpu::StepLoggingConfig exit_log_cfg(model);
    {
        exit_log_cfg.agent("Circle").logMean<float>("totalMessageCount");
    }

    // Create the simulation object
    flamegpu::CUDASimulation simulation(model);

#ifdef VISUALISATION
    flamegpu::visualiser::ModelVis& visualisation = simulation.getVisualisation();
    {
        visualisation.setInitialCameraLocation(ENV_WIDTH, ENV_WIDTH, ENV_WIDTH);
        visualisation.setInitialCameraTarget(0.0f, 0.0f, 0.0f);
        visualisation.setCameraSpeed(0.001f * ENV_WIDTH);
        visualisation.setViewClips(0.1f, 5000);
        visualisation.setClearColor(1.0f, 1.0f, 1.0f);
        visualisation.setFPSColor(0.0f, 0.0f, 0.0f);
        visualisation.setBeginPaused(true);
        auto& agt = visualisation.addAgent("Circle");
        agt.setModel(flamegpu::visualiser::Stock::Models::SPHERE);
        agt.setModelScale(0.1f);
    }
    visualisation.activate();
#endif


    // Set config configuraiton properties 
    simulation.SimulationConfig().timing = false;
    simulation.SimulationConfig().verbose = false;
    simulation.SimulationConfig().random_seed = runInputs.SEED;
    simulation.SimulationConfig().steps = runInputs.STEPS;
    simulation.CUDAConfig().device_id = runInputs.CUDA_DEVICE;

    // Generate the initial population
    std::mt19937_64 rng(runInputs.SEED);
    std::uniform_real_distribution<float> dist(ENV_MIN, ENV_MAX);
    flamegpu::AgentVector population(model.Agent("Circle"), runInputs.AGENT_COUNT);
    for (unsigned int i = 0; i < runInputs.AGENT_COUNT; i++) {
        flamegpu::AgentVector::Agent instance = population[i];
        instance.setVariable<int>("id", i);
        instance.setVariable<float>("x", dist(rng));
        instance.setVariable<float>("y", dist(rng));
        instance.setVariable<float>("z", dist(rng));
    }

    // Set the population for the simulation.
    simulation.setPopulationData(population);

    //attach loggin configs
    simulation.setStepLog(step_log_cfg);
    simulation.setExitLog(exit_log_cfg);

    // Execute 
    simulation.simulate();

#ifdef VISUALISATION
    visualisation.join();
#endif

    // get step log data
    runOutputs.drift_per_step = std::make_shared<std::vector<double>>();
    runOutputs.messages_per_step = std::make_shared<std::vector<double>>();
    flamegpu::RunLog run_log = simulation.getRunLog();
    std::list<flamegpu::LogFrame> step_log = run_log.getStepLog();
    for (auto& log : step_log) {
        runOutputs.drift_per_step->push_back(log.getAgent("Circle").getMean("drift"));
        runOutputs.messages_per_step->push_back(log.getAgent("Circle").getMean("stepMessageCount"));
    }

    // get timing data
    std::vector<double> s_steps = simulation.getElapsedTimeSteps();
    runOutputs.s_per_step = std::make_shared<std::vector<double>>(s_steps.begin(), s_steps.end());
    runOutputs.s_stepMean = std::accumulate(s_steps.begin(), s_steps.end(), 0.0) / simulation.getStepCounter();

    // get message count from exit log
    flamegpu::LogFrame exit_log = run_log.getExitLog();
    runOutputs.mean_messageCount = exit_log.getAgent("Circle").getMean("totalMessageCount") / (double)runInputs.STEPS;
}
